#include<cstdio>

int main(void) {
    hipDeviceProp_t prop;

    int count;
    hipGetDeviceCount(&count);
    for (int i = 0;i < count;i++) {
        hipGetDeviceProperties(&prop, i);
        printf("   --- General Information for device %d ---\n", i);
        printf("Name:  %s\n", prop.name);
        printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
        printf("Clock rate:  %d\n", prop.clockRate);
    }
}